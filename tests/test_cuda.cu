
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel
__global__ void vectorAdd(int *a, int *b, int *c, int size) {
    // block idx is block number, blockDim.x is the dimension of each block
    // tid is the index in the array
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // you need this check because the array may not divide evenly
    // ex: 1000 => 256, 256, 256, 256 => has extra threads
    if(tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

int testSimpleCuda() {
    std::cout << "Hello, World!" << std::endl;

    int size = 1024;
    int* vecA = new int[size];
    int* vecB = new int[size];
    int* vecC = new int[size];

    for(int i = 0; i < size; ++i) {
        vecA[i] = i;
        vecB[i] = 2 * i;
    }

    int *vecACuda, *vecBCuda, *vecCCuda;
    hipMalloc(&vecACuda, size * sizeof(int));
    hipMalloc(&vecBCuda, size * sizeof(int));
    hipMalloc(&vecCCuda, size * sizeof(int));

    hipMemcpy(vecACuda, vecA, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vecBCuda, vecB, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(vecCCuda, vecC, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // blocking call to cuda
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(vecACuda, vecBCuda, vecCCuda, size);

    hipMemcpy(vecC, vecCCuda, size * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < size; ++i) {
        if(vecC[i] != vecA[i] + vecB[i]) {
            std::cerr << "incorrect value at index " << i << std::endl;
            break;
        }
        std::cout << vecA[i] << " " << vecB[i] << " " << vecC[i] << std::endl;
    }

    delete[] vecA;
    delete[] vecB;
    delete[] vecC;
    hipFree(vecACuda);
    hipFree(vecBCuda);
    hipFree(vecCCuda);

    return 0;
}
